#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime_api.h>
#include<assert.h>
#include<typeinfo> 

struct matrix{
	int n_x;
	void* data;

	matrix(int nx){

		n_x = nx;
		data = (void*)malloc(nx*sizeof(double));

	}

}



__global__  void add(double *a, double *b, double *c){

	c[blockIdx.x] = a[blockIdx.x] + c[blockIdx.x];
}



void fill_array(double *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}



void print_array(matrix &a )
{

for(int idx = 0; idx< a.n_x; idx++)
	std::cout<<a.data[idx]<<std::endl;


}





int main() {
		int N = 550;

		matrix a(N);
		matrix b(N);
		matrix c(N);

		matrix d_a(N);
		matrix d_b(N);
		matrix d_c(N);

		fill_array((double*) a.data);
		fill_array((double*) b.data);
		std::cout<<a.data[5]<<std::endl;

        // Alloc space for device copies of a, b, c
        hipError_t result1 = hipMalloc(&d_a.data, d_a.n_x * sizeof(double));
        hipError_t result2 = hipMalloc(&d_b.data, d_b.n_x * sizeof(double));
        hipError_t result3 = hipMalloc(&d_c.data, d_c.n_x * sizeof(double));
		assert(result1 == hipSuccess || result2 == hipSuccess || result3 == hipSuccess);



       // Copy inputs to device
        result1 = hipMemcpy(&d_a.data, &a.data, d_a.n_x * sizeof(double), hipMemcpyHostToDevice);
        result2 = hipMemcpy(&d_b.data, &b.data, d_b.n_x * sizeof(double), hipMemcpyHostToDevice);
		assert(result1 == hipSuccess || result2 == hipSuccess);


		add<<<N,1>>> add((double*)d_a.data,(double*)d_b.data,(double*)d_c.data )


        // Copy result back to host
        hipError_t result = hipMemcpy(&c.data, &d_c.data, d_a.n_x * sizeof(double), hipMemcpyDeviceToHost);
		assert(result == hipSuccess);





        hipFree(d_a.data); hipFree(d_b.data); hipFree(d_c.data);



	return 0;
}
