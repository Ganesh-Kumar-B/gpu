
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime_api.h>
#include<assert.h>
#include<typeinfo> 


template<typename T>
struct matrix{

	int n_x;
	void* data;

	
	matrix(int nx){

		n_x = nx;
		data = (void*)malloc(nx*sizeof(T));

	}

}
;

template<typename T>
__global__ void device_add(T *a, T *b, T *c) {

        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}



template<typename T>
void fill_array(matrix<T> &a) {
	T *ptr = (T*)a.data;
	for(int idx=0;idx<a.n_x;idx++)
		ptr[idx] = idx;
}


template<typename T>
void print_array(matrix<T> &a )
{
T *ptr = (T*)a.data;
for(int idx = 0; idx< a.n_x; idx++)
	std::cout<<ptr[idx]<<std::endl;


}





int main() {
		int N = 50;

		matrix<double> a(N);
		matrix<double> b(N);
		matrix<double> c(N);

		matrix<double> d_a(N);
		matrix<double> d_b(N);
		matrix<double> d_c(N);

		fill_array(a);
		fill_array(b);
		std::cout<<*((double*)a.data +5 )<<std::endl;

        // Alloc space for device copies of a, b, c
        hipError_t result1 = hipMalloc(&d_a.data, d_a.n_x * sizeof(double));
        hipError_t result2 = hipMalloc(&d_b.data, d_b.n_x * sizeof(double));
        hipError_t result3 = hipMalloc(&d_c.data, d_c.n_x * sizeof(double));
	assert(result1 == hipSuccess || result2 == hipSuccess || result3 == hipSuccess);



       // Copy inputs to device
        result1 = hipMemcpy(d_a.data, a.data, d_a.n_x * sizeof(double), hipMemcpyHostToDevice);
        result2 = hipMemcpy(d_b.data, b.data, d_b.n_x * sizeof(double), hipMemcpyHostToDevice);
	assert(result1 == hipSuccess || result2 == hipSuccess);

	device_add<<<1,N>>>((double*)d_a.data,(double*)d_b.data, (double*)d_c.data);  

        // Copy result back to host
        hipError_t result = hipMemcpy(c.data, d_c.data, d_a.n_x * sizeof(double), hipMemcpyDeviceToHost);
	assert(result == hipSuccess);



	print_array(c);

        hipFree(d_a.data); hipFree(d_b.data); hipFree(d_c.data);



	return 0;
}

