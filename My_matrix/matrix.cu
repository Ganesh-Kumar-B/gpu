
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime_api.h>
#include<assert.h>
#include<typeinfo> 











template<typename T>
struct matrix{

	int n_x;
	T* data;

	
	matrix(int nx){

		n_x = nx;
		data = (T*)malloc(nx*sizeof(T));

	}

}
;

template<typename T>
__global__ void device_add(T *a, T *b, T *c) {

        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}



template<typename T>
void fill_array(matrix<T> &a) {
	
	for(int idx=0;idx<a.n_x;idx++)
		a.data[idx] = idx;
}


template<typename T>
void print_array(matrix<T> &a )
{
	for(int idx = 0; idx< a.n_x; idx++)
		std::cout<<a.data[idx]<<std::endl;


}





int main() {
		int N = 500;

		matrix<double> a(N);
		matrix<double> b(N);
		matrix<double> c(N);

		matrix<double> d_a(N);
		matrix<double> d_b(N);
		matrix<double> d_c(N);

		fill_array(a);
		fill_array(b);
		std::cout<<a.data[5]<<std::endl;

        // Alloc space for device copies of a, b, c
        hipError_t result1 = hipMalloc((void**)&d_a.data, d_a.n_x * sizeof(a.data[0]));
        hipError_t result2 = hipMalloc((void**)&d_b.data, d_b.n_x * sizeof(b.data[0]));
        hipError_t result3 = hipMalloc((void**)&d_c.data, d_c.n_x * sizeof(c.data[0]));
	assert(result1 == hipSuccess || result2 == hipSuccess || result3 == hipSuccess);



       // Copy inputs to device
        result1 = hipMemcpy((void*)d_a.data,(void*) a.data, d_a.n_x * sizeof(d_a.data[0]), hipMemcpyHostToDevice);
        result2 = hipMemcpy((void*)d_b.data,(void*) b.data, d_b.n_x * sizeof(d_b.data[0]), hipMemcpyHostToDevice);
	assert(result1 == hipSuccess || result2 == hipSuccess);

	device_add<<<1,N>>>(d_a.data,d_b.data, d_c.data);  

        // Copy result back to host
        hipError_t result = hipMemcpy((void*)c.data, (void*)d_c.data, d_a.n_x * sizeof(d_c.data[0]), hipMemcpyDeviceToHost);
	assert(result == hipSuccess);



	print_array(c);

        hipFree(d_a.data); hipFree(d_b.data); hipFree(d_c.data);



	return 0;
}












































// template<typename T>
// struct matrix{

// 	int n_x;
// 	void* data;

	
// 	matrix(int nx){

// 		n_x = nx;
// 		data = (void*)malloc(nx*sizeof(T));

// 	}

// }
// ;

// template<typename T>
// __global__ void device_add(T *a, T *b, T *c) {

//         c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
// }



// template<typename T>
// void fill_array(matrix<T> &a) {
// 	T *ptr = (T*)a.data;
// 	for(int idx=0;idx<a.n_x;idx++)
// 		ptr[idx] = idx;
// }


// template<typename T>
// void print_array(matrix<T> &a )
// {
// T *ptr = (T*)a.data;
// for(int idx = 0; idx< a.n_x; idx++)
// 	std::cout<<ptr[idx]<<std::endl;


// }





// int main() {
// 		int N = 500;

// 		matrix<double> a(N);
// 		matrix<double> b(N);
// 		matrix<double> c(N);

// 		matrix<double> d_a(N);
// 		matrix<double> d_b(N);
// 		matrix<double> d_c(N);

// 		fill_array(a);
// 		fill_array(b);
// 		std::cout<<*((double*)a.data +5 )<<std::endl;

//         // Alloc space for device copies of a, b, c
//         cudaError_t result1 = cudaMalloc(&d_a.data, d_a.n_x * sizeof(double));
//         cudaError_t result2 = cudaMalloc(&d_b.data, d_b.n_x * sizeof(double));
//         cudaError_t result3 = cudaMalloc(&d_c.data, d_c.n_x * sizeof(double));
// 	assert(result1 == cudaSuccess || result2 == cudaSuccess || result3 == cudaSuccess);



//        // Copy inputs to device
//         result1 = cudaMemcpy(d_a.data, a.data, d_a.n_x * sizeof(double), cudaMemcpyHostToDevice);
//         result2 = cudaMemcpy(d_b.data, b.data, d_b.n_x * sizeof(double), cudaMemcpyHostToDevice);
// 	assert(result1 == cudaSuccess || result2 == cudaSuccess);

// 	device_add<<<1,N>>>((double*)d_a.data,(double*)d_b.data, (double*)d_c.data);  

//         // Copy result back to host
//         cudaError_t result = cudaMemcpy(c.data, d_c.data, d_a.n_x * sizeof(double), cudaMemcpyDeviceToHost);
// 	assert(result == cudaSuccess);



// 	print_array(c);

//         cudaFree(d_a.data); cudaFree(d_b.data); cudaFree(d_c.data);



// 	return 0;
// }

